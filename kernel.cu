//
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <algorithm>
//Limite de hilos 1024

#define M 4096

__global__ void filtroKernel(unsigned char *a)
{
	int j=blockIdx.x;
	int i=threadIdx.x;
	int R,G,B;
	int tr,tg,tb;
	for(i=threadIdx.x;i<4096;i+=1024)
	{
		
		if(i>1||i<4094||j>1||j<4094)
		{
			tr=tg=tb=0;
			R=i+j*4096;
			G=i+j*4096+4096*4096;
			B=i+j*4096+4096*4096*2;
			for(int x=-2;x<3;x++)
			{
				for(int y=-2;y<3;y++)
				{
					tr+=a[R+x+y*M];
					tg+=a[G+x+y*M];
					tb+=a[B+x+y*M];
				}
			}
			__syncthreads:
			a[R]=tr/25;
			a[G]=tg/25;
			a[B]=tb/25;
		}
	}

}

__global__ void bordeKernel(unsigned char* a,int* temp)
{
	int j=blockIdx.x;
	int i=threadIdx.x;
	int R,G,B;

	if(j==0||j==4095)
		return;
	
	for(i=threadIdx.x;i<4095;i+=1024)
	{
		if(i==0)
			continue;
		R=i+j*4096;
		G=i+j*4096+4096*4096;
		B=i+j*4096+4096*4096*2;

		for(int x=-1;x<2;x++)
		{
			for(int y=-1;y<2;y++)
			{
				temp[R]+=std::abs(a[R]-a[R+x+M*y]);
				temp[R]+=std::abs(a[G]-a[G+x+M*y]);
				temp[R]+=std::abs(a[B]-a[B+x+M*y]);
			}
		}

		temp[R]/=9;
	}





	
}

__global__ void finKernel(unsigned char* a,int* temp)
{
	int j=blockIdx.x;
	int i=threadIdx.x;
	int R,G,B;
	for(i=threadIdx.x;i<4096;i+=1024)
	{
		R=i+j*4096;
		G=i+j*4096+4096*4096;
		B=i+j*4096+4096*4096*2;
		//a[R]=a[G]=a[B]=temp[R];
		
		
		if(temp[R]>20)
		{
			a[R]=a[G]=a[B]=255;
		}
		else
		{
			a[R]=a[G]=a[B]=0;
		}
		
	}
}

extern "C"
int main2(unsigned char* imag)
{
	hipError_t cudaStatus=hipSetDevice(0);
	unsigned char *Im=0;
	int *Temp=0;
	cudaStatus = hipMalloc<unsigned char>(&Im,4096*4096*3*sizeof(unsigned char));
	
	cudaStatus = hipMalloc<int>(&Temp,4096*4096*sizeof(int));
	
	cudaStatus = hipMemcpy(Im,imag,4096*4096*3*sizeof(unsigned char),hipMemcpyHostToDevice);

	cudaStatus= hipMemset(Temp,0,4096*4096*sizeof(int));
	
	filtroKernel<<<4096,1024>>>(Im);

	hipDeviceSynchronize();
	bordeKernel<<<4096,1024>>>(Im,Temp);
	cudaStatus=hipGetLastError();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1! fallo lanzamiento!  Lo has hecho bien?\n");
    }
	hipDeviceSynchronize();
	finKernel<<<4096,1024>>>(Im,Temp);



	cudaStatus=hipMemcpy(imag,Im,4096*4096*3*sizeof(unsigned char),hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2! fallo lanzamiento!  Lo has hecho bien?\n");
    }

    return 0;
}
/*
// Helper function for using CUDA to add vectors in parallel.
cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    cudaError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = cudaSetDevice(0);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = cudaGetLastError();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = cudaDeviceSynchronize();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

Error:
    cudaFree(dev_c);
    cudaFree(dev_a);
    cudaFree(dev_b);
    
    return cudaStatus;
}
*/